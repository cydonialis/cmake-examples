#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void add_one(int* data, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx] += 1;
    }
}

int main(){
    const int N = 8;
    int h_data[N] = {0, 1, 2, 3, 4, 5, 6, 7};
    int *d_data;

    hipMalloc((void**)&d_data, N * sizeof(int));

    hipMemcpy(d_data, h_data, N * sizeof(int), hipMemcpyHostToDevice);

    add_one<<<1, N>>>(d_data, N);

    hipMemcpy(h_data, d_data, N * sizeof(int), hipMemcpyDeviceToHost);

    for(auto item: h_data) {
        cout << item << " ";
    }
    cout << endl;
    return 0;
}
